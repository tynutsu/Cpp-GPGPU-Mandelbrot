#include "hip/hip_runtime.h"
#include <cmath>
#include <limits>

#include "MandelbrotSet.h"

using uchar = unsigned char;


#ifndef TOTAL_SHADES
#define TOTAL_SHADES 16
#endif

const uchar maxit = std::numeric_limits<uchar>::max();

MandelbrotSet *set;

struct Complex { double x, y; };

__constant__ Pixel shades[TOTAL_SHADES] =
{ { 66,30,15 },{ 25,7,26 },{ 9,1,47 },{ 4,4,73 },{ 0,7,100 },
{ 12,44,138 },{ 24,82,177 },{ 57,125,209 },{ 134,181,229 },{ 211,236,248 },
{ 241,233,191 },{ 248,201,95 },{ 255,170,0 },{ 204,128,0 },{ 153,87,0 },
{ 106,52,3 } };

__global__ void calc_mandel(Pixel  *img_data, const int width, const int height, const double scale)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int index = row * width + col;
	float x0 = ((float)col / width) * 3.5f - 2.5f;
	float y0 = ((float)row / height) * 3.5f - 1.75f;

	float x = 0.0f;
	float y = 0.0f;
	int iter = 0;
	float xtemp;
	while ((x * x + y * y <= 4.0f) && (iter < maxit))
	{
		xtemp = x * x - y * y + x0;
		y = 2 * x * y + y0;
		x = xtemp;
		iter++;
	}
	if (iter == maxit || iter == 0) {
		img_data[index].r = 0; img_data[index].g = 0; img_data[index].b = 0;
	}
	else {
		img_data[index] = shades[iter % TOTAL_SHADES];
	}	
}

 
void process(MandelbrotSet* set, double scale) {
	dim3 block_size(16, 16);
	int w = set->getWidth();
	int h = set->getHeight();
	dim3 grid_size(w / block_size.x, h / block_size.y);	
	calc_mandel << <grid_size, block_size >> >(set->getDeviceReference(), w, h, scale);
	set->saveAs("testOutput.ppm");
}

int main(int argc, char *argv[])
{
  const int width  = (argc > 1) ? std::atoi(argv[1]) : 4096;
  const int height = (argc > 2) ? std::atoi(argv[2]) : 4096;
  const double scale = 1. / (width / 4);
  set = new MandelbrotSet(width, height);
  for (int i = 0; i < 5; i++) {
	  cout << "Attempt [" << i << "] " << endl;
	  measure(process, set, scale);
  }
  delete set;
  //std::cin.get();
  return 0;
}
